
#include <hip/hip_runtime_api.h>
#include "cudaOpenMP.h"
#include "matlabData.h"

void CudaOpenMPQMMD::evolution_with_coriolis_2(const double dt, const int calculate_energy)
{
  const int &n1 = MatlabData::r1()->n;
  const int &n2 = MatlabData::r2()->n;
  const int &n_legs = MatlabData::omega_states()->l_max;
  const int &n_theta = MatlabData::theta()->n; 
  insist(n_theta >= n_legs);
  const size_t n = n1*n2*n_legs;
  
#pragma omp parallel for default(shared)
  for(int i_dev = 0; i_dev < n_gpus(); i_dev++) {
    omega_wavepackets_on_single_device[i_dev]->zero_coriolis_variables();
  }

  setup_streams_and_events(1, 1);
  
  for(int i_dev = 0; i_dev < n_gpus(); i_dev++) {
    
    OmegaWavepacketsOnSingleDevice * &omega_wpts_i_dev = omega_wavepackets_on_single_device[i_dev];
    const Vec<OmegaWavepacket *> &omega_wpts_i = omega_wpts_i_dev->omega_wavepackets;

    for(int i = 0; i < omega_wpts_i.size(); i++) {
      
      const Complex *psi_i = omega_wpts_i[i]->legendre_psi_dev_pointer();
      
      for(int j_dev = 0; j_dev < n_gpus(); j_dev++) {
	
	if(i_dev != j_dev) {
	  
	  OmegaWavepacketsOnSingleDevice * &omega_wpts_j_dev = omega_wavepackets_on_single_device[j_dev];
	  
	  omega_wpts_j_dev->zero_work_dev_2(&streams[j_dev]);
	  checkCudaErrors(hipEventRecord(events[j_dev], streams[j_dev]));
	  
	  checkCudaErrors(hipStreamWaitEvent(streams_p2p[i_dev], events[j_dev], 0)); 
	  checkCudaErrors(hipSetDevice(i_dev));
	  checkCudaErrors(hipMemcpyPeerAsync(omega_wpts_j_dev->work_dev_2, j_dev,
					      psi_i, i_dev, n*sizeof(Complex), streams_p2p[i_dev]));
	  checkCudaErrors(hipEventRecord(events[i_dev], streams_p2p[i_dev]));
	  
	  checkCudaErrors(hipStreamWaitEvent(streams[j_dev], events[i_dev], 0)); 
	  
	  omega_wpts_j_dev->evolution_with_coriolis(dt, omega_wpts_i[i]->omega_value(),
						    omega_wpts_j_dev->work_dev_2, &streams[j_dev]);
	  if(calculate_energy)
	    omega_wpts_j_dev->calculate_coriolis_energy_for_legendre_psi(omega_wpts_i[i]->omega_value(),
									 omega_wpts_j_dev->work_dev_2, 
									 &streams[j_dev]);
	} else {
	  omega_wpts_i_dev->evolution_with_coriolis(dt, omega_wpts_i[i]->omega_value(),
						    psi_i, &streams[i_dev]);
	  if(calculate_energy)
	    omega_wpts_i_dev->calculate_coriolis_energy_for_legendre_psi(omega_wpts_i[i]->omega_value(),
									 psi_i, &streams[i_dev]);
	}
      }
    }
  }

  update_evolution_with_coriolis();
}
