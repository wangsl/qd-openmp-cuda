
#include <hip/hip_runtime_api.h>
#include "cudaOpenMP.h"
#include "matlabData.h"

void CudaOpenMPQMMD::evolution_with_coriolis_with_p2p(const double dt, const int calculate_energy)
{
  if(n_gpus() == 1) return;
  
  const int &n1 = MatlabData::r1()->n;
  const int &n2 = MatlabData::r2()->n;
  const int &n_legs = MatlabData::omega_states()->l_max;
  const int &n_theta = MatlabData::theta()->n; 
  insist(n_theta >= n_legs);
  const size_t n = n1*n2*n_legs;
  
  for(int i_dev = 0; i_dev < n_gpus(); i_dev++) {
    
    OmegaWavepacketsOnSingleDevice * &omega_wpts_i_dev = omega_wavepackets_on_single_device[i_dev];
    const Vec<OmegaWavepacket *> &omega_wpts_i = omega_wpts_i_dev->omega_wavepackets;
    
    for(int i = 0; i < omega_wpts_i.size(); i++) {
      
      const Complex *psi_i = omega_wpts_i[i]->legendre_psi_dev_pointer();

      for(int j_dev = 0; j_dev < n_gpus(); j_dev++) {
	
	if(i_dev == j_dev) continue;
	
	OmegaWavepacketsOnSingleDevice * &omega_wpts_j_dev = omega_wavepackets_on_single_device[j_dev];
	
	omega_wpts_j_dev->zero_work_dev_2();
	checkCudaErrors(hipMemcpyPeer(omega_wpts_j_dev->work_dev_2, j_dev,
				       psi_i, i_dev, n*sizeof(Complex)));
	
	const int &omega_i = omega_wpts_i[i]->omega_value();
	omega_wpts_j_dev->evolution_with_coriolis(dt, omega_i, omega_wpts_j_dev->work_dev_2);
	
	if(calculate_energy) 
	  omega_wpts_j_dev->calculate_coriolis_energy_for_legendre_psi(omega_i,
								       omega_wpts_j_dev->work_dev_2);
      }
    }
  }
}

void CudaOpenMPQMMD::evolution_with_coriolis_with_p2p_async(const double dt, 
							    const int calculate_energy)
{
  if(n_gpus() == 1) return;
  
  const int &n1 = MatlabData::r1()->n;
  const int &n2 = MatlabData::r2()->n;
  const int &n_legs = MatlabData::omega_states()->l_max;
  const int &n_theta = MatlabData::theta()->n; 
  insist(n_theta >= n_legs);
  const size_t n = n1*n2*n_legs;

  setup_streams_and_events(1, 0);
  
  for(int i_dev = 0; i_dev < n_gpus(); i_dev++) {
    
    OmegaWavepacketsOnSingleDevice * &omega_wpts_i_dev = omega_wavepackets_on_single_device[i_dev];
    const Vec<OmegaWavepacket *> &omega_wpts_i = omega_wpts_i_dev->omega_wavepackets;
    
    for(int i = 0; i < omega_wpts_i.size(); i++) {
      
      const Complex *psi_i = omega_wpts_i[i]->legendre_psi_dev_pointer();
      
      for(int j_dev = 0; j_dev < n_gpus(); j_dev++) {
	
	if(i_dev == j_dev) continue;
	
	OmegaWavepacketsOnSingleDevice * &omega_wpts_j_dev = omega_wavepackets_on_single_device[j_dev];

	omega_wpts_j_dev->zero_work_dev_2();
	checkCudaErrors(hipMemcpyPeer(omega_wpts_j_dev->work_dev_2, j_dev,
				       psi_i, i_dev, n*sizeof(Complex)));

	omega_wpts_j_dev->evolution_with_coriolis(dt, omega_wpts_i[i]->omega_value(),
						  omega_wpts_j_dev->work_dev_2, &streams[j_dev]);

	if(calculate_energy) {
	  omega_wpts_j_dev->calculate_coriolis_energy_for_legendre_psi(omega_wpts_i[i]->omega_value(),
								       omega_wpts_j_dev->work_dev_2, 
								       &streams[j_dev]);
	}
      }
    }
  }
}
