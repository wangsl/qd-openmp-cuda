#include "hip/hip_runtime.h"

#include "cudaMath.h"
#include "evolutionUtils.h"

static __global__ void _evolution_with_potential_(Complex *psi, const double *pot, 
						  const int n, const double dt)
{
  const int index = threadIdx.x + blockDim.x*blockIdx.x;
  if (index < n) 
    psi[index] *= exp(Complex(0.0, -dt)*pot[index]);
}

static __global__ void _evolution_with_kinetic_(Complex *psi, const int n1, const int n2, 
						const int nLeg, const double dt)
{
  extern __shared__ double s_data[];
  
  double *kin1 = (double *) s_data;
  double *kin2 = (double *) &kin1[n1];
  
  cudaMath::setup_kinetic_energy_for_fft(kin1, r1_dev.n, r1_dev.n*r1_dev.dr, r1_dev.mass);
  cudaMath::setup_kinetic_energy_for_fft(kin2, r2_dev.n, r2_dev.n*r2_dev.dr, r2_dev.mass);
  __syncthreads();
  
  const int index = threadIdx.x + blockDim.x*blockIdx.x;
  if(index < n1*n2*nLeg) {
    int i = -1; int j = -1; int l = -1;
    cudaMath::index_2_ijk(index, n1, n2, nLeg, i, j, l);
    psi[index] *= exp(Complex(0.0, -dt)*(kin1[i] + kin2[j]));
  }
}

static __global__ void _evolution_with_rotational_(Complex *psi, const int n1, const int n2, 
						   const int nLegs, const int omega, const double dt)
{
  extern __shared__ double s_data[];
  
  double *I1 = (double *) s_data;
  double *I2 = (double *) &I1[n1];
  
  cudaMath::setup_moments_of_inertia(I1, r1_dev.n, r1_dev.left, r1_dev.dr, r1_dev.mass);
  cudaMath::setup_moments_of_inertia(I2, r2_dev.n, r2_dev.left, r2_dev.dr, r2_dev.mass);

  __syncthreads();
  
  const int index = threadIdx.x + blockDim.x*blockIdx.x;
  
  if(index < n1*n2*nLegs) {
    int i = -1; int j = -1; int l = -1;
    cudaMath::index_2_ijk(index, n1, n2, nLegs, i, j, l);
    l += omega;
    psi[index] *= exp(Complex(0.0, -dt)*l*(l+1)*(I1[i] + I2[j]));
  }
}

static __global__ void _psi_times_kinitic_energy_(Complex *psi_out, const Complex *psi_in,
                                                  const int n1, const int n2)
{
  extern __shared__ double s_data[];
  
  double *kin1 = (double *) s_data;
  double *kin2 = (double *) &kin1[n1];
  
  cudaMath::setup_kinetic_energy_for_fft(kin1, r1_dev.n, r1_dev.n*r1_dev.dr, r1_dev.mass);
  cudaMath::setup_kinetic_energy_for_fft(kin2, r2_dev.n, r2_dev.n*r2_dev.dr, r2_dev.mass);
  __syncthreads();
  
  const int index = threadIdx.x + blockDim.x*blockIdx.x;
  if(index < n1*n2) {
    int i = -1; int j = -1;
    cudaMath::index_2_ij(index, n1, n2, i, j);
    psi_out[index] = psi_in[index]*(kin1[i] + kin2[j]);
  }
}

static __global__ void _psi_times_moments_of_inertia_(Complex *psi_out, const Complex *psi_in, 
						      const int n1, const int n2)
{
  extern __shared__ double s_data[];
  
  double *I1 = (double *) s_data;
  double *I2 = (double *) &I1[n1];
  
  cudaMath::setup_moments_of_inertia(I1, r1_dev.n, r1_dev.left, r1_dev.dr, r1_dev.mass);
  cudaMath::setup_moments_of_inertia(I2, r2_dev.n, r2_dev.left, r2_dev.dr, r2_dev.mass);
 
  __syncthreads();
  
  const int index = threadIdx.x + blockDim.x*blockIdx.x;
  if(index < n1*n2) {
    int i = -1; int j = -1;
    cudaMath::index_2_ij(index, n1, n2, i, j);
    psi_out[index] = (I1[i] + I2[j])*psi_in[index];
  }
}

static __global__ void _print_coriolis_on_device_(const int n, const double *e, const double *v)
{
  if(e) {
    for(int i = 0; i < n; i++)
      printf("%18.12f", e[i]);
    printf("\n");
  }
  
  for(int i = 0; i < n; i++) {
    for(int j = 0; j < n; j++) { 
      const int k = cudaMath::ij_2_index(n, n, i, j);
      printf("%18.12f", v[k]);
    }
    printf("\n");
  }
}

static __global__ void _calculate_coriolis_on_device_(const int n, const double *e, 
						      const double *v, double *b)
{
  for(int i = 0; i < n; i++) {
    for(int j = 0; j < n; j++) {
      const int ij = cudaMath::ij_2_index(n, n, i, j);
      b[ij] = 0;
      for(int alpha = 0; alpha < n; alpha++) {
	const int i_alpha = cudaMath::ij_2_index(n, n, i, alpha);
	const int j_alpha = cudaMath::ij_2_index(n, n, j, alpha);
	b[ij] += e[alpha]*v[i_alpha]*v[j_alpha];
      }
    }
  }
}

static __global__ void _calculate_coriolis_on_device_(const int n, 
						      const double *e, const double *v, 
						      const int omega, const int omega1, 
						      double *b)
{
  const int index = threadIdx.x + blockDim.x*blockIdx.x;
  
  if(index < r1_dev.n) {
    
    const double R = r1_dev.left + index*r1_dev.dr;
    const double I = 2*r1_dev.mass*R*R;
    
    const int &i = omega;
    const int &j = omega1;
    
    b[index] = 0.0;
    for(int alpha = 0; alpha < n; alpha++) {
      const int i_alpha = cudaMath::ij_2_index(n, n, i, alpha);
      const int j_alpha = cudaMath::ij_2_index(n, n, j, alpha);
      b[index] += -e[alpha]/I*v[i_alpha]*v[j_alpha];
    }
  }
}

static __global__ void _calculate_coriolis_on_device_(const int n, 
						      const double *e, const double *v, 
						      const int omega, const int omega1, 
						      Complex *b)
{
  const int index = threadIdx.x + blockDim.x*blockIdx.x;
  
  if(index < r1_dev.n) {
    
    const double R = r1_dev.left + index*r1_dev.dr;
    const double I = 2*r1_dev.mass*R*R;
    
    const int &i = omega;
    const int &j = omega1;
    
    b[index].zero();
    for(int alpha = 0; alpha < n; alpha++) {
      const int i_alpha = cudaMath::ij_2_index(n, n, i, alpha);
      const int j_alpha = cudaMath::ij_2_index(n, n, j, alpha);
      b[index] += Complex(-e[alpha]/I*v[i_alpha]*v[j_alpha], 0.0);
    }
  }
}

static __global__ void _evolution_with_coriolis_(Complex *psi_out, const int n1, const int n2,
						 const double *e, const double *v, const int n, 
						 const int omega, const int omega1, const double dt, 
						 const Complex *psi_in)
{
  extern __shared__ Complex expC[];
  
  for(int i = threadIdx.x; i < n1; i += blockDim.x) {
    
    const double R = r1_dev.left + i*r1_dev.dr;
    const double I = 2*r1_dev.mass*R*R;
    
    const Complex dt_I(0.0, dt/I);
    
    expC[i].zero();
    for(int alpha = 0; alpha < n; alpha++) {
      const int omega_alpha = cudaMath::ij_2_index(n, n, omega, alpha);
      const int omega1_alpha = cudaMath::ij_2_index(n, n, omega1, alpha);
      expC[i] += exp(-dt_I*e[alpha])*v[omega_alpha]*v[omega1_alpha];
    }
  }
  
  __syncthreads();
  
  const int index = threadIdx.x + blockDim.x*blockIdx.x;
  
  if(index < n1*n2) {
    int i = -1; int j = -1;
    cudaMath::index_2_ij(index, n1, n2, i, j);
    psi_out[index] += expC[i]*psi_in[index];
  }
}

static __global__ void _coriolis_matrices_production_(Complex *psi_out, const Complex *psi_in,
						      const int n1, const int n2,
						      const double *e, const double *v, const int n, 
						      const int omega, const int omega1)
{
  extern __shared__ double c[];
  
  for(int i = threadIdx.x; i < n1; i += blockDim.x) {
    
    c[i] = 0.0;
    for(int alpha = 0; alpha < n; alpha++) {
      const int omega_alpha = cudaMath::ij_2_index(n, n, omega, alpha);
      const int omega1_alpha = cudaMath::ij_2_index(n, n, omega1, alpha);
      c[i] += e[alpha]*v[omega_alpha]*v[omega1_alpha];
    }
    
    const double R = r1_dev.left + i*r1_dev.dr;
    const double I = 2*r1_dev.mass*R*R;
    c[i] /= I;
  }
  
  __syncthreads();
  
  const int index = threadIdx.x + blockDim.x*blockIdx.x;
  
  if(index < n1*n2) {
    int i = -1; int j = -1;
    cudaMath::index_2_ij(index, n1, n2, i, j);
    psi_out[index] = c[i]*psi_in[index];
  }
}

static __global__ void _dump_wavepacket_(Complex *psi, const int n1, const int n2, const int n_theta)
{
  const int index = threadIdx.x + blockDim.x*blockIdx.x;

  if(index < n1*n2*n_theta) {
    int i = -1; int j = -1; int k = -1;
    cudaMath::index_2_ijk(index, n1, n2, n_theta, i, j, k);
    psi[index] *= dump1_dev[i]*dump2_dev[j];
  }
}
