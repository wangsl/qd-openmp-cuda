
#include <hip/hip_runtime_api.h>
#include "omegawavepacket.h"
#include "deviceConst.h"
#include "cudaUtils.h"
#include "cudaMath.h"

#include "evolutionCUDAaux.cu"

/*
  legendre_psi_dev dimesnions 0..l_max (lmax+1)
  real data is legendre_psi_dev(:, :, omega..lmax)
  but for FFT, we'll have to use all data
*/

OmegaWavepacket::OmegaWavepacket(const int &omega_,
				 const int &l_max_,
				 const Vec<CoriolisMatrixAux> &coriolis_matrices_,
				 const RMat &associated_legendres_,
				 const RadialCoordinate &r1_,
				 const RadialCoordinate &r2_,
				 const AngleCoordinate &theta_,
				 Complex *psi_, 
				 const double *pot_dev_,
				 hipblasHandle_t &cublas_handle_,
				 hipfftHandle &cufft_plan_for_legendre_psi_,
				 Complex * &work_dev_) :
  omega(omega_), l_max(l_max_), coriolis_matrices(coriolis_matrices_),
  associated_legendres(associated_legendres_),
  r1(r1_), r2(r2_), theta(theta_),
  psi(psi_), 
  pot_dev(pot_dev_), psi_dev(0),
  legendre_psi_dev(0), associated_legendres_dev(0), 
  weighted_associated_legendres_dev(0),
  cublas_handle(cublas_handle_),
  cufft_plan_for_legendre_psi(cufft_plan_for_legendre_psi_),
  work_dev(work_dev_),
  _wavepacket_module(0), _potential_energy(0), _kinetic_energy(0), 
  _rotational_energy(0), _coriolis_energy(0),
  _wavepacket_module_for_legendre_psi(0)
{ 
  insist(theta.n > l_max+1);

  insist(psi);
  insist(work_dev);
  setup_device_data();
}

OmegaWavepacket::~OmegaWavepacket() 
{
  std::cout << " Destruct OmegaWavepacket: omega = " << omega << std::endl;
  psi = 0;
  pot_dev = 0;
  
  _CUDA_FREE_(psi_dev);
  _CUDA_FREE_(legendre_psi_dev);
  _CUDA_FREE_(associated_legendres_dev);
  _CUDA_FREE_(weighted_associated_legendres_dev);
}

void OmegaWavepacket::setup_device_data()
{
  std::cout << " Setup OmegaWavepacket: omega = " << omega << " l_max = "<< l_max << std::endl;
  
  copy_psi_from_host_to_device();
  setup_associated_legendres();
  setup_weighted_associated_legendres();
  setup_legendre_psi();
}

void OmegaWavepacket::calculate_wavepacket_module()
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;
  
  const double *w = theta.w;
  
  hipStream_t *streams = (hipStream_t *) malloc(n_theta*sizeof(hipStream_t));
  insist(streams);
  for(int k = 0; k < n_theta; k++) 
    checkCudaErrors(hipStreamCreate(&streams[k]));
  
  Complex *dots = new Complex [n_theta];
  insist(dots);
  memset(dots, 0, sizeof(Complex)*n_theta);
  
  const hipDoubleComplex *psi_ = (hipDoubleComplex *) psi_dev;
  
  for(int k = 0; k < n_theta; k++) {
    insist(hipblasSetStream(cublas_handle, streams[k]) == HIPBLAS_STATUS_SUCCESS);

    insist(hipblasZdotc(cublas_handle, n1*n2, psi_, 1, psi_, 1, (hipDoubleComplex *) &dots[k]) ==
	   HIPBLAS_STATUS_SUCCESS);
    
    psi_ += n1*n2;
  }
  
  checkCudaErrors(hipDeviceSynchronize());
  
  double &sum = _wavepacket_module;
  sum = 0.0;
  for(int k = 0; k < n_theta; k++)
    sum += w[k]*dots[k].real();
  
  sum *= r1.dr*r2.dr;
  
  if(dots) { delete [] dots; dots = 0; }
  
  for(int k = 0; k < n_theta; k++) 
    checkCudaErrors(hipStreamDestroy(streams[k]));
  
  if(streams) { free(streams); streams = 0; }
  
  hipblasSetStream(cublas_handle, 0);
}

void OmegaWavepacket::calculate_potential_energy()
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;
  
  const double *w = theta.w;
  
  insist(work_dev);
  hipDoubleComplex *psi_tmp_ = (hipDoubleComplex *) work_dev;
  
  const int n_threads = _NTHREADS_;
  const int n_blocks = cudaUtils::number_of_blocks(n_threads, n1*n2);
  
  double &sum = _potential_energy;
  sum = 0.0;
  for(int k = 0; k < n_theta; k++) {
    
    const hipDoubleComplex *psi_ = (hipDoubleComplex *) (psi_dev + k*n1*n2);
    
    cudaMath::_vector_multiplication_<Complex, Complex, double><<<n_blocks, n_threads>>>
      ((Complex *) psi_tmp_, (const Complex *) psi_, pot_dev+k*n1*n2, n1*n2);
    
    Complex dot(0.0, 0.0);
    insist(hipblasZdotc(cublas_handle, n1*n2, psi_, 1, psi_tmp_, 1, (hipDoubleComplex *) &dot) ==
	   HIPBLAS_STATUS_SUCCESS);
    
    sum += w[k]*dot.real();
  }
  
  sum *= r1.dr*r2.dr;
}

void OmegaWavepacket::setup_associated_legendres()
{
  if(associated_legendres_dev) return;
  
  const int &n_theta = theta.n;
  const int n_legs = l_max - omega + 1;
  insist(n_legs > 0);
  
  const RMat &p = associated_legendres;
  insist(p.rows() == n_theta);
  
  Mat<Complex> p_complex(n_legs, n_theta);
  for(int l = 0; l < n_legs; l++) {
    for(int k = 0; k < n_theta; k++) {
      p_complex(l,k) = Complex(p(k,l), 0.0);
    }
  }
  
  std::cout << " Allocate device memory for complex associated Legendre Polynomials: " 
            << n_legs << " " << n_theta << std::endl;
  
  const int size = n_legs*n_theta;
  checkCudaErrors(hipMalloc(&associated_legendres_dev, size*sizeof(Complex)));
  checkCudaErrors(hipMemcpyAsync(associated_legendres_dev, (const Complex *) p_complex,
				  size*sizeof(Complex), hipMemcpyHostToDevice));
}

void OmegaWavepacket::setup_weighted_associated_legendres()
{
  if(weighted_associated_legendres_dev) return;
  
  const int &n_theta = theta.n;
  const int n_legs = l_max - omega + 1;
  insist(n_legs > 0);
  
  const double *w = theta.w;
  
  const RMat &p = associated_legendres;
  insist(p.rows() == n_theta);
  
  Mat<Complex> wp_complex(n_theta, n_legs);
  for(int l = 0; l < n_legs; l++) {
    for(int k = 0; k < n_theta; k++) {
      wp_complex(k,l) = Complex(w[k]*p(k,l), 0.0);
    }
  }
  
  std::cout << " Allocate device memory for weighted complex associated Legendre Polynomials: " 
            << n_theta << " " << n_legs << std::endl;
  
  const int size = n_theta*n_legs;
  checkCudaErrors(hipMalloc(&weighted_associated_legendres_dev, size*sizeof(Complex)));
  checkCudaErrors(hipMemcpyAsync(weighted_associated_legendres_dev, (const Complex *) wp_complex,
				  size*sizeof(Complex), hipMemcpyHostToDevice));
}

void OmegaWavepacket::setup_legendre_psi()
{
  if(legendre_psi_dev) return;
  
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int n_legs = l_max + 1;
  
  std::cout << " Allocate device memory for Legendre psi: "
            << n1 << " " << n2 << " " << n_legs << std::endl;
  
  const size_t size = n1*n2*n_legs;
  
  checkCudaErrors(hipMalloc(&legendre_psi_dev, size*sizeof(Complex)));
  insist(legendre_psi_dev);
  checkCudaErrors(hipMemset(legendre_psi_dev, 0, size*sizeof(Complex)));
}

void OmegaWavepacket::forward_legendre_transform()
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;
  const int n_legs = l_max - omega + 1;
  
  const Complex one(1.0, 0.0);
  const Complex zero(0.0, 0.0);

  Complex *legendre_psi_dev_ = legendre_psi_dev + omega*n1*n2;
  
  insist(hipblasZgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                     n1*n2, n_legs, n_theta, 
                     (const hipDoubleComplex *) &one,
                     (const hipDoubleComplex *) psi_dev, n1*n2,
                     (const hipDoubleComplex *) weighted_associated_legendres_dev, n_theta,
                     (const hipDoubleComplex *) &zero,
                     (hipDoubleComplex *) legendre_psi_dev_, n1*n2) == HIPBLAS_STATUS_SUCCESS);
}

void OmegaWavepacket::backward_legendre_transform()
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;
  const int n_legs = l_max - omega + 1;
  
  const Complex one(1.0, 0.0);
  const Complex zero(0.0, 0.0);

  Complex *legendre_psi_dev_ = legendre_psi_dev + omega*n1*n2;

  insist(hipblasZgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                     n1*n2, n_theta, n_legs,
                     (const hipDoubleComplex *) &one,
                     (const hipDoubleComplex *) legendre_psi_dev_, n1*n2,
                     (const hipDoubleComplex *) associated_legendres_dev, n_legs,
                     (const hipDoubleComplex *) &zero,
                     (hipDoubleComplex *) psi_dev, n1*n2) == HIPBLAS_STATUS_SUCCESS);
}

void OmegaWavepacket::forward_fft_for_legendre_psi()
{ 
  insist(hipfftExecZ2Z(cufft_plan_for_legendre_psi, (hipDoubleComplex *) legendre_psi_dev,
                      (hipDoubleComplex *) legendre_psi_dev, HIPFFT_FORWARD) == HIPFFT_SUCCESS);
}

void OmegaWavepacket::backward_fft_for_legendre_psi(const int do_scale)
{
  insist(hipfftExecZ2Z(cufft_plan_for_legendre_psi, (hipDoubleComplex *) legendre_psi_dev, 
                      (hipDoubleComplex *) legendre_psi_dev, HIPFFT_BACKWARD) == HIPFFT_SUCCESS);
  
  if(do_scale) {
    const int &n1 = r1.n;
    const int &n2 = r2.n;
    const int n_legs = l_max + 1;
    
    const double s = 1.0/(n1*n2);
    insist(hipblasZdscal(cublas_handle, n1*n2*n_legs, &s, (hipDoubleComplex *) legendre_psi_dev, 1) 
           == HIPBLAS_STATUS_SUCCESS);
  }
}

void OmegaWavepacket::copy_psi_from_device_to_host()
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;
  
  insist(psi && psi_dev);
  checkCudaErrors(hipMemcpyAsync(psi, psi_dev, n1*n2*n_theta*sizeof(Complex), hipMemcpyDeviceToHost));
}

void OmegaWavepacket::copy_psi_from_host_to_device()
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;
  
  if(!psi_dev) 
    checkCudaErrors(hipMalloc(&psi_dev, n1*n2*n_theta*sizeof(Complex)));
  insist(psi_dev);
  
  checkCudaErrors(hipMemcpyAsync(psi_dev, psi, n1*n2*n_theta*sizeof(Complex), hipMemcpyHostToDevice));
}


void OmegaWavepacket::calculate_wavepacket_module_for_legendre_psi()
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int n_legs = l_max - omega + 1;
  
  const Complex *legendre_psi_dev_ = legendre_psi_dev + omega*n1*n2;
  
  Complex s(0.0, 0.0);
  
  insist(hipblasZdotc(cublas_handle, n1*n2*n_legs,
		     (const hipDoubleComplex *) legendre_psi_dev_, 1, 
		     (const hipDoubleComplex *) legendre_psi_dev_, 1,
		     (hipDoubleComplex *) &s) == HIPBLAS_STATUS_SUCCESS);
  
  _wavepacket_module_for_legendre_psi = s.real()*r1.dr*r2.dr;
}

void OmegaWavepacket::evolution_with_potential(const double dt)
{
  insist(pot_dev && psi_dev);
  
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;
  const int n = n1*n2*n_theta;
  
  const int n_threads = _NTHREADS_;
  const int n_blocks = cudaUtils::number_of_blocks(n_threads, n);
  
  _evolution_with_potential_<<<n_blocks, n_threads>>>(psi_dev, pot_dev, n, dt);
}

void OmegaWavepacket::evolution_with_kinetic(const double dt)
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int n_legs = l_max - omega + 1;
  
  const int n = n1*n2*n_legs;

  Complex *legendre_psi_dev_ = legendre_psi_dev + n1*n2*omega;
  
  const int n_threads = _NTHREADS_;
  const int n_blocks = cudaUtils::number_of_blocks(n_threads, n);
  
  _evolution_with_kinetic_<<<n_blocks, n_threads, (n1+n2)*sizeof(double)>>>
    (legendre_psi_dev_, n1, n2, n_legs, dt);
}

void OmegaWavepacket::evolution_with_rotational(const double dt)
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int n_legs = l_max - omega + 1;
  
  const int n = n1*n2*n_legs;

  Complex *legendre_psi_dev_ = legendre_psi_dev + n1*n2*omega;
  
  const int n_threads = _NTHREADS_;
  const int n_blocks = cudaUtils::number_of_blocks(n_threads, n);
  
  _evolution_with_rotational_<<<n_blocks, n_threads, (n1+n2)*sizeof(double)>>>
    (legendre_psi_dev_, n1, n2, n_legs, dt);
}

void OmegaWavepacket::calculate_kinetic_energy_for_legendre_psi()
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int n_legs = l_max - omega + 1;
  
  insist(work_dev);
  hipDoubleComplex *psi_tmp_dev = (hipDoubleComplex *) work_dev;
  
  const int n_threads = _NTHREADS_;
  const int n_blocks = cudaUtils::number_of_blocks(n_threads, n1*n2);
  
  double sum = 0.0;
  for(int l = 0; l < n_legs; l++) {
    
    const hipDoubleComplex *legendre_psi_dev_ = (hipDoubleComplex *) legendre_psi_dev + n1*n2*(l+omega);
    
    _psi_times_kinitic_energy_<<<n_blocks, n_threads, (n1+n2)*sizeof(double)>>>
      ((Complex *) psi_tmp_dev, (const Complex *) legendre_psi_dev_, n1, n2);
    
    Complex dot(0.0, 0.0);
    insist(hipblasZdotc(cublas_handle, n1*n2, legendre_psi_dev_, 1, psi_tmp_dev, 1, 
                       (hipDoubleComplex *) &dot) == HIPBLAS_STATUS_SUCCESS);
    
    sum += dot.real();
  }
  
  sum *= r1.dr*r2.dr/n1/n2;
  
  _kinetic_energy = sum;
}

void OmegaWavepacket::calculate_rotational_energy_for_legendre_psi()
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int n_legs = l_max - omega + 1;
  
  insist(work_dev);
  hipDoubleComplex *psi_tmp_dev = (hipDoubleComplex *) work_dev;

  const int n_threads = _NTHREADS_;
  const int n_blocks = cudaUtils::number_of_blocks(n_threads, n1*n2);
  
  double sum = 0.0;
  for(int l = 0; l < n_legs; l++) {
    const hipDoubleComplex *legendre_psi_in_dev = (hipDoubleComplex *) legendre_psi_dev + n1*n2*(l+omega);
    
    _psi_times_moments_of_inertia_<<<n_blocks, n_threads, (n1+n2)*sizeof(double)>>>
      ((Complex *) psi_tmp_dev, (const Complex *) legendre_psi_in_dev, n1, n2);
    
    Complex dot(0.0, 0.0);
    insist(hipblasZdotc(cublas_handle, n1*n2, legendre_psi_in_dev, 1, psi_tmp_dev, 1, 
                       (hipDoubleComplex *) &dot) == HIPBLAS_STATUS_SUCCESS);
    
    sum += (l+omega)*(l+omega+1)*dot.real();
  }
  
  sum *= r1.dr*r2.dr;
  
  _rotational_energy = sum;
}

void OmegaWavepacket::evolution_with_coriolis(const double dt, 
					      const int l, const int omega1,
					      const double *coriolis_matrices_dev,
					      const Complex *legendre_psi_omega1,
					      hipStream_t *stream)
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;

  Complex *legendre_psi_dev_ = psi_dev + l*n1*n2;;
  
  const int n = coriolis_matrices[l].omega_max - coriolis_matrices[l].omega_min + 1;
  const double *e = coriolis_matrices_dev + coriolis_matrices[l].offset;
  const double *v = e + n;
  
  const int n_threads = _NTHREADS_;
  const int n_blocks = cudaUtils::number_of_blocks(n_threads, n1*n2);

  if(stream) 
    _evolution_with_coriolis_<<<n_blocks, n_threads, n1*sizeof(Complex), *stream>>>
      (legendre_psi_dev_, n1, n2, e, v, n, omega, omega1, dt, legendre_psi_omega1);
  else
    _evolution_with_coriolis_<<<n_blocks, n_threads, n1*sizeof(Complex)>>>
      (legendre_psi_dev_, n1, n2, e, v, n, omega, omega1, dt, legendre_psi_omega1);
  
}

void OmegaWavepacket::evolution_with_coriolis(const double dt, const int omega1,
					      const double *coriolis_matrices_dev,
					      const Complex *legendre_psi_omega1)
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;

  const int n_streams = l_max + 1;
  
  hipStream_t *streams = (hipStream_t *) malloc(n_streams*sizeof(hipStream_t));
  insist(streams);
  for(int i = 0; i < n_streams; i++) 
    checkCudaErrors(hipStreamCreate(&streams[i]));
  
  int i_stream = 0;
  for(int l = 0; l <= l_max; l++) {
    if(coriolis_matrices[l].l == -1) continue;
    
    evolution_with_coriolis(dt, l, omega1, coriolis_matrices_dev,
			    legendre_psi_omega1+l*n1*n2, &streams[i_stream]);
    i_stream++;
  }
  insist(i_stream <= n_streams);
  
  checkCudaErrors(hipDeviceSynchronize());
  
  for(int i = 0; i < n_streams; i++) 
    checkCudaErrors(hipStreamDestroy(streams[i]));
  if(streams) { free(streams); streams = 0; }
}

void OmegaWavepacket::zero_psi_dev()
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;
  const int n = n1*n2*n_theta;
  checkCudaErrors(hipMemset(psi_dev, 0, n*sizeof(Complex)));
}

void OmegaWavepacket::update_evolution_with_coriolis()
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int n_legs = l_max - omega + 1;
  
  checkCudaErrors(hipMemcpy(legendre_psi_dev + n1*n2*omega, psi_dev + n1*n2*omega, 
			     n1*n2*n_legs*sizeof(Complex), hipMemcpyDeviceToDevice));
}

void OmegaWavepacket::calculate_coriolis_energy_for_legendre_psi(const int omega1,
								 const double *coriolis_matrices_dev,
								 const Complex *legendre_psi_omega1)
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  
  Complex *C = 0;
  checkCudaErrors(hipMalloc(&C, n1*(l_max+1)*sizeof(Complex)));
  checkCudaErrors(hipMemset(C, 0, n1*(l_max+1)*sizeof(Complex)));
  insist(C);
  
  Complex *psi_dot_tmp = 0;
  checkCudaErrors(hipMalloc(&psi_dot_tmp, n1*(l_max+1)*sizeof(Complex)));
  checkCudaErrors(hipMemset(psi_dot_tmp, 0, n1*(l_max+1)*sizeof(Complex)));

  for(int l = 0; l <= l_max; l++) {
    
    if(coriolis_matrices[l].l == -1) continue;
    
    const int n = coriolis_matrices[l].omega_max - coriolis_matrices[l].omega_min + 1;
    const double *e = coriolis_matrices_dev + coriolis_matrices[l].offset;
    const double *v = e + n;
    
    const int n_threads = _NTHREADS_;
    const int n_blocks = cudaUtils::number_of_blocks(n_threads, n1);  
    _calculate_coriolis_on_device_<<<n_blocks, n_threads>>>(n, e, v, omega, omega1, C + l*n1);
  }
  
  insist(hipblasSetPointerMode(cublas_handle, HIPBLAS_POINTER_MODE_DEVICE) == HIPBLAS_STATUS_SUCCESS);

  for(int l = 0; l <= l_max; l++) {
    
    if(coriolis_matrices[l].l == -1) continue;
    
    for(int i = 0; i < n1; i++) {
      
      const Complex *legendre_psi_dev_ = legendre_psi_dev + l*n1*n2 + i;
      const Complex *legendre_psi_omega1_ = legendre_psi_omega1 + l*n1*n2 + i;
      
      const int k = cudaMath::ij_2_index(n1, l_max+1, i, l);

      insist(hipblasZdotc(cublas_handle, n2, 
			 (const hipDoubleComplex *) legendre_psi_dev_, n1, 
			 (const hipDoubleComplex *) legendre_psi_omega1_, n1, 
			 (hipDoubleComplex *) psi_dot_tmp+k ) == HIPBLAS_STATUS_SUCCESS);
    }
  }

  checkCudaErrors(hipDeviceSynchronize());
  
  insist(hipblasSetPointerMode(cublas_handle, HIPBLAS_POINTER_MODE_HOST) == HIPBLAS_STATUS_SUCCESS);
  
  insist(hipblasSetStream(cublas_handle, NULL) == HIPBLAS_STATUS_SUCCESS);

  Complex dot(0.0, 0.0);
  
  insist(hipblasZdotu(cublas_handle, n1*(l_max+1), 
		     (const hipDoubleComplex *) C, 1, 
		     (const hipDoubleComplex *) psi_dot_tmp, 1, 
		     (hipDoubleComplex *) &dot) == HIPBLAS_STATUS_SUCCESS);

  _coriolis_energy += dot.real()*r1.dr*r2.dr;
    
  _CUDA_FREE_(C);
  _CUDA_FREE_(psi_dot_tmp);
}


void OmegaWavepacket::calculate_coriolis_energy_for_legendre_psi_2(const int omega1,
								   const double *coriolis_matrices_dev,
								   const Complex *legendre_psi_omega1)
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int n_legs = l_max + 1;

  double *C = 0;
  checkCudaErrors(hipMalloc(&C, n1*(l_max+1)*sizeof(double)));
  checkCudaErrors(hipMemset(C, 0, n1*(l_max+1)*sizeof(double)));
  insist(C);
  
  for(int l = 0; l <= l_max; l++) {
    
    if(coriolis_matrices[l].l == -1) continue;
    
    const int n = coriolis_matrices[l].omega_max - coriolis_matrices[l].omega_min + 1;
    const double *e = coriolis_matrices_dev + coriolis_matrices[l].offset;
    const double *v = e + n;
    
    const int n_threads = _NTHREADS_;
    const int n_blocks = cudaUtils::number_of_blocks(n_threads, n1);  
    _calculate_coriolis_on_device_<<<n_blocks, n_threads>>>(n, e, v, omega, omega1, C + l*n1);
  }

  Complex *tmp = 0;
  checkCudaErrors(hipMalloc(&tmp, n1*n2*(l_max+1)*sizeof(Complex)));
  insist(tmp);
  
  const int n_threads = _NTHREADS_;
  const int n_blocks = cudaUtils::number_of_blocks(n_threads, n1*n2*n_legs); 
  
  _coriolis_matrices_production_<<<n_blocks, n_threads>>>(legendre_psi_omega1, C, tmp, n1, n2, n_legs);
  
  Complex dot(0.0, 0.0);
  
  insist(hipblasZdotc(cublas_handle, n1*n2*n_legs, 
		     (const hipDoubleComplex *) legendre_psi_dev, 1, 
		     (const hipDoubleComplex *) tmp, 1, 
		     (hipDoubleComplex *) &dot) == HIPBLAS_STATUS_SUCCESS);
  
  if(omega != omega1)
    dot *= 2;
  
  _coriolis_energy += dot.real()*r1.dr*r2.dr;
  
  _CUDA_FREE_(C);
  _CUDA_FREE_(tmp);
}
