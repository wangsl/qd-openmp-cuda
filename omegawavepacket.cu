#include "hip/hip_runtime.h"

#include <hip/hip_runtime_api.h>
#include "omegawavepacket.h"
#include "deviceConst.h"
#include "cudaUtils.h"
#include "cudaMath.h"

#include "evolutionCUDAaux.cu"

OmegaWavepacket::OmegaWavepacket(const int &omega_,
				 const int &l_max_,
				 const RMat &associated_legendres_,
				 const RadialCoordinate &r1_,
				 const RadialCoordinate &r2_,
				 const AngleCoordinate &theta_,
				 Complex *psi_, 
				 const double *pot_dev_,
				 hipblasHandle_t &cublas_handle_,
				 hipfftHandle &cufft_plan_for_legendre_psi_,
				 Complex * &work_dev_) :
  omega(omega_), l_max(l_max_),
  associated_legendres(associated_legendres_),
  r1(r1_), r2(r2_), theta(theta_),
  psi(psi_), 
  pot_dev(pot_dev_), psi_dev(0),
  legendre_psi_dev(0), associated_legendres_dev(0), 
  weighted_associated_legendres_dev(0),
  cublas_handle(cublas_handle_),
  cufft_plan_for_legendre_psi(cufft_plan_for_legendre_psi_),
  work_dev(work_dev_),
  _wavepacket_module(0),
  _potential_energy(0),
  _wavepacket_module_for_legendre_psi(0)
{ 
  insist(psi);
  insist(work_dev);
  setup_device_data();
}

OmegaWavepacket::~OmegaWavepacket() 
{
  std::cout << " Destruct OmegaWavepacket: omega = " << omega << std::endl;
  psi = 0;
  pot_dev = 0;
  
  _CUDA_FREE_(psi_dev);
  _CUDA_FREE_(legendre_psi_dev);
  _CUDA_FREE_(associated_legendres_dev);
  _CUDA_FREE_(weighted_associated_legendres_dev);
}

void OmegaWavepacket::setup_device_data()
{
  std::cout << " Setup OmegaWavepacket: omega = " << omega << " " << work_dev << " " << l_max << std::endl;
  
  copy_psi_from_host_to_device();
  setup_associated_legendres();
  setup_weighted_associated_legendres();
  setup_legendre_psi();
}

void OmegaWavepacket::calculate_wavepacket_module()
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;
  
  const double *w = theta.w;
  
  hipStream_t *streams = (hipStream_t *) malloc(n_theta*sizeof(hipStream_t));
  for(int k = 0; k < n_theta; k++) 
    checkCudaErrors(hipStreamCreate(&streams[k]));
  
  Complex *dots = new Complex [n_theta];
  insist(dots);
  memset(dots, 0, sizeof(Complex)*n_theta);
  
  const hipDoubleComplex *psi_ = (hipDoubleComplex *) psi_dev;
  
  for(int k = 0; k < n_theta; k++) {
    
    insist(hipblasSetStream(cublas_handle, streams[k]) == HIPBLAS_STATUS_SUCCESS);
    
    insist(hipblasZdotc(cublas_handle, n1*n2, psi_, 1, psi_, 1, (hipDoubleComplex *) &dots[k]) ==
	   HIPBLAS_STATUS_SUCCESS);
    
    psi_ += n1*n2;
  }
  
  checkCudaErrors(hipDeviceSynchronize());
  
  double &sum = _wavepacket_module;
  sum = 0.0;
  for(int k = 0; k < n_theta; k++)
    sum += w[k]*dots[k].real();
  
  sum *= r1.dr*r2.dr;
  
  if(dots) { delete [] dots; dots = 0; }
  
  for(int k = 0; k < n_theta; k++) 
    checkCudaErrors(hipStreamDestroy(streams[k]));
  
  if(streams) { free(streams); streams = 0; }
  
  hipblasSetStream(cublas_handle, 0);
}

void OmegaWavepacket::calculate_potential_energy()
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;
  
  const double *w = theta.w;
  
  insist(work_dev);
  hipDoubleComplex *psi_tmp_ = (hipDoubleComplex *) work_dev;
  
  const int n_threads = _NTHREADS_;
  const int n_blocks = cudaUtils::number_of_blocks(n_threads, n1*n2);
  
  double &sum = _potential_energy;
  sum = 0.0;
  for(int k = 0; k < n_theta; k++) {

    const hipDoubleComplex *psi_ = (hipDoubleComplex *) (psi_dev + k*n1*n2);
    
    cudaMath::_vector_multiplication_<Complex, Complex, double><<<n_blocks, n_threads>>>
      ((Complex *) psi_tmp_, (const Complex *) psi_, pot_dev+k*n1*n2, n1*n2);
    
    Complex dot(0.0, 0.0);
    insist(hipblasZdotc(cublas_handle, n1*n2, psi_, 1, psi_tmp_, 1, (hipDoubleComplex *) &dot) ==
	   HIPBLAS_STATUS_SUCCESS);
    
    sum += w[k]*dot.real();
  }
  
  sum *= r1.dr*r2.dr;
}

void OmegaWavepacket::setup_associated_legendres()
{
  if(associated_legendres_dev) return;
  
  const int &n_theta = theta.n;
  const int n_legs = l_max - omega + 1;
  insist(n_legs > 0);
  
  const RMat &p = associated_legendres;
  insist(p.rows() == n_theta);
  
  Mat<Complex> p_complex(n_legs, n_theta);
  for(int l = 0; l < n_legs; l++) {
    for(int k = 0; k < n_theta; k++) {
      p_complex(l,k) = Complex(p(k,l), 0.0);
    }
  }
  
  std::cout << " Allocate device memory for complex associated Legendre Polynomials: " 
            << n_legs << " " << n_theta << std::endl;
  
  const int size = n_legs*n_theta;
  checkCudaErrors(hipMalloc(&associated_legendres_dev, size*sizeof(Complex)));
  checkCudaErrors(hipMemcpyAsync(associated_legendres_dev, (const Complex *) p_complex,
				  size*sizeof(Complex), hipMemcpyHostToDevice));
}

void OmegaWavepacket::setup_weighted_associated_legendres()
{
  if(weighted_associated_legendres_dev) return;
  
  const int &n_theta = theta.n;
  const int n_legs = l_max - omega + 1;
  insist(n_legs > 0);
  
  const double *w = theta.w;
  
  const RMat &p = associated_legendres;
  insist(p.rows() == n_theta);
  
  Mat<Complex> wp_complex(n_theta, n_legs);
  for(int l = 0; l < n_legs; l++) {
    for(int k = 0; k < n_theta; k++) {
      wp_complex(k,l) = Complex(w[k]*p(k,l), 0.0);
    }
  }
  
  std::cout << " Allocate device memory for weighted complex associated Legendre Polynomials: " 
            << n_theta << " " << n_legs << std::endl;
  
  const int size = n_theta*n_legs;
  checkCudaErrors(hipMalloc(&weighted_associated_legendres_dev, size*sizeof(Complex)));
  checkCudaErrors(hipMemcpyAsync(weighted_associated_legendres_dev, (const Complex *) wp_complex,
				  size*sizeof(Complex), hipMemcpyHostToDevice));
}

void OmegaWavepacket::setup_legendre_psi()
{
  if(legendre_psi_dev) return;
  
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  //const int n_legs = std::max(l_max+1, theta.n);
  const int n_legs = l_max + 1;
  
  std::cout << " Allocate device memory for Legendre psi: "
            << n1 << " " << n2 << " " << n_legs << std::endl;
  
  const size_t size = n1*n2*n_legs;
  
  checkCudaErrors(hipMalloc(&legendre_psi_dev, size*sizeof(Complex)));
  insist(legendre_psi_dev);
  checkCudaErrors(hipMemset(legendre_psi_dev, 0, size*sizeof(Complex)));
}

void OmegaWavepacket::forward_legendre_transform()
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;
  const int n_legs = l_max - omega + 1;
  
  const Complex one(1.0, 0.0);
  const Complex zero(0.0, 0.0);

  Complex *legendre_psi_dev_ = legendre_psi_dev + omega*n1*n2;
  
  insist(hipblasZgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                     n1*n2, n_legs, n_theta, 
                     (const hipDoubleComplex *) &one,
                     (const hipDoubleComplex *) psi_dev, n1*n2,
                     (const hipDoubleComplex *) weighted_associated_legendres_dev, n_theta,
                     (const hipDoubleComplex *) &zero,
                     (hipDoubleComplex *) legendre_psi_dev_, n1*n2) == HIPBLAS_STATUS_SUCCESS);
}

void OmegaWavepacket::backward_legendre_transform()
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;
  const int n_legs = l_max - omega + 1;
  
  const Complex one(1.0, 0.0);
  const Complex zero(0.0, 0.0);

  Complex *legendre_psi_dev_ = legendre_psi_dev + omega*n1*n2;

  insist(hipblasZgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
                     n1*n2, n_theta, n_legs,
                     (const hipDoubleComplex *) &one,
                     (const hipDoubleComplex *) legendre_psi_dev_, n1*n2,
                     (const hipDoubleComplex *) associated_legendres_dev, n_legs,
                     (const hipDoubleComplex *) &zero,
                     (hipDoubleComplex *) psi_dev, n1*n2) == HIPBLAS_STATUS_SUCCESS);
}

void OmegaWavepacket::forward_fft_for_legendre_psi()
{ 
  insist(hipfftExecZ2Z(cufft_plan_for_legendre_psi, (hipDoubleComplex *) legendre_psi_dev,
                      (hipDoubleComplex *) legendre_psi_dev, HIPFFT_FORWARD) == HIPFFT_SUCCESS);
}

void OmegaWavepacket::backward_fft_for_legendre_psi(const int do_scale)
{
  insist(hipfftExecZ2Z(cufft_plan_for_legendre_psi, (hipDoubleComplex *) legendre_psi_dev, 
                      (hipDoubleComplex *) legendre_psi_dev, HIPFFT_BACKWARD) == HIPFFT_SUCCESS);
  
  if(do_scale) {
    const int &n1 = r1.n;
    const int &n2 = r2.n;
    const int n_legs = l_max + 1;
    
    const double s = 1.0/(n1*n2);
    insist(hipblasZdscal(cublas_handle, n1*n2*n_legs, &s, (hipDoubleComplex *) legendre_psi_dev, 1) 
           == HIPBLAS_STATUS_SUCCESS);
  }
}

void OmegaWavepacket::copy_psi_from_device_to_host()
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;
  
  insist(psi && psi_dev);
  checkCudaErrors(hipMemcpyAsync(psi, psi_dev, n1*n2*n_theta*sizeof(Complex), hipMemcpyDeviceToHost));
}

void OmegaWavepacket::copy_psi_from_host_to_device()
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int &n_theta = theta.n;
  
  if(!psi_dev) 
    checkCudaErrors(hipMalloc(&psi_dev, n1*n2*n_theta*sizeof(Complex)));

  insist(psi_dev);
  checkCudaErrors(hipMemcpyAsync(psi_dev, psi, n1*n2*n_theta*sizeof(Complex), hipMemcpyHostToDevice));
}


void OmegaWavepacket::calculate_wavepacket_module_for_legendre_psi()
{
  const int &n1 = r1.n;
  const int &n2 = r2.n;
  const int n_legs = l_max - omega + 1;

  const Complex *legendre_psi_dev_ = legendre_psi_dev + omega*n1*n2;
  
  Complex s(0.0, 0.0);

  insist(hipblasZdotc(cublas_handle, n1*n2*n_legs,
		     (const hipDoubleComplex *) legendre_psi_dev_, 1, 
		     (const hipDoubleComplex *) legendre_psi_dev_, 1,
		     (hipDoubleComplex *) &s) == HIPBLAS_STATUS_SUCCESS);
  
  _wavepacket_module_for_legendre_psi = s.real()*r1.dr*r2.dr;
}
